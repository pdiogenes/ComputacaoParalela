/*
Tempo sequencial:
    real    1m6.463s
    user    1m6.338s
    sys     0m0.068s

Tempo paralelo CPU:
    real    0m36.188s
    user    2m21.357s
    sys     0m3.156s

Tempo paralelo GPU:
    real    0m21.687s
    user    0m17.254s
    sys     0m4.460s

Tempo paralelo GPU CUDA:
    real    0m1.498s
    user    0m0.515s
    sys     0m0.890s



--

Métricas do nvprof:


    distribute parallel for simd:

    Invocations                                Event Name         Min Max Avg
Total Device "GeForce GT 1030 (0)" Kernel: mm$_omp_fn$0 1 warps_launched 72 72
72          72

    ==28418== Metric result:
    Invocations                               Metric Name Metric Description Min
Max         Avg Device "GeForce GT 1030 (0)" Kernel: mm$_omp_fn$0 1
warp_execution_efficiency                 Warp Execution Efficiency      90.31%
90.31%      90.31%


    CUDA gpu:
    Invocations                                Event Name         Min         Max         Avg       Total
    Device "GeForce GT 1030 (0)"
        Kernel: mm(double*, double*, double*, int)
            1                            warps_launched      127008      127008      127008      127008

    ==26255== Metric result:
    Invocations                               Metric Name                        Metric Description         Min         Max         Avg
    Device "GeForce GT 1030 (0)"
        Kernel: mm(double*, double*, double*, int)
            1                 warp_execution_efficiency                 Warp Execution Efficiency      99.21%      99.21%      99.21%




*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mm(double* a, double* b, double* c, int width) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < width && j < width){
        double sum = 0;
        for (int k = 0; k < width; k++) {
            double x = a[i * width + k];
            double y = b[k * width + j];
            sum += x * y;
        }
        c[i * width + j] = sum;
    }
}

int main() {
    int width = 2000;
    int size = width * width * sizeof(double);
    double* a = (double*)malloc(size);
    double* b = (double*)malloc(size);
    double* c = (double*)malloc(size);

    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            a[i * width + j] = i;
            b[i * width + j] = j;
            c[i * width + j] = 0;
        }
    }

    double *d_a, *d_b, *d_c;

    hipMalloc((void **) &d_a, size);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    hipMalloc((void **) &d_b, size);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    hipMalloc((void **) &d_c, size);

    int block_size = 32;
    int blocks = (width-1)/block_size + 1;
    dim3 dimGrid(blocks, blocks, 1);
    dim3 dimBlock(block_size, block_size, 1);

    mm<<<dimGrid,dimBlock>>>(d_a, d_b, d_c, width);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            printf("\n c[%d][%d] = %f", i, j, c[i * width + j]);
        }
    }
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}